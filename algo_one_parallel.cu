#define _CRT_SECURE_NO_DEPRECATE

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
// #define size 1024

#define CUDA_WARN(XXX) \
    do {if (XXX != hipSuccess) printf("%s\n", hipGetErrorString(XXX));} while (0)

#define BLOCK_ROW 32
#define BLOCK_COL 32

__global__ void kernal_process_image(float* image, int height, int width, int kernel)
{
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    if (row>0 && col>0 && row<height-1 && col<width-1 && image[row*width+col]==0)
    {
        float div=0, su=0, wei=0;
        int i, j;
        for(i=-1; i<=1; i++)
            for(j=-1; j<=1; j++)
                div+=image[(row+i)*width+col+j];

        if(div>0.06)
        {
            int range = kernel/2;
            for(i=-range; i<=range; i++){
                for(j=-range;j<=range;j++){
                    if(row+i<0 || row+i>height || col+j<0 || col+j>width || image[(row+i)*width+col+j]==0){
                        continue;
                    }
                    wei += 1 / sqrt((i*i) + (j*j));
                    su += (image[(row+i)*width+col+j]/sqrt((i*i) + (j*j)));
                }
            }
        }
        image[row*width+col]=(wei!=0)?su/wei:0;
        // printf("%d,", row*width+col);
    }
    return;
}

void process_image(float* input_image, int height, int width)
{
    float* image;

    hipMalloc(&image, height*width*sizeof(float));

    hipMemcpy(image, input_image, height*width*sizeof(float), hipMemcpyHostToDevice);

    dim3 dimBlock(BLOCK_ROW, BLOCK_COL);
    dim3 dimGrid((height-1)/dimBlock.x + 1, (width-1)/dimBlock.y + 1);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    kernal_process_image <<<dimBlock, dimGrid>>> (image, height, width, 5);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float elapsed_time=0;
    hipEventElapsedTime(&elapsed_time, start, stop);

    printf("Time to identify black pixels and replace those with weighted average for image size : %d x %d is: %f miliseconds\n",height,width,elapsed_time);

    hipMemcpy(input_image, image, height*width*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(image);
    return;
}


int main(int argc, char **argv)
{
    int i, j, x;
	clock_t start,end;
	unsigned char byte[54];

    if(argc<3)
    {
        printf("Insufficient input Argument");
        return 1;
    }

	start = clock();

	FILE* fIn = fopen(argv[1], "rb");//Input File name
	FILE* fOut = fopen(argv[2], "wb");//Output File name


	if (fIn == NULL)											// check if the input file has not been opened succesfully.
	{
		printf("File does not exist.\n");
	}

	end = clock();
	double walltime=(double)(((double)(end-start)* 1000)/(double)CLOCKS_PER_SEC);
    printf("Time to open input & output image is: %f miliseconds\n",walltime);


    start = clock();
	for (i = 0; i < 54; i++)											//read the 54 byte header from fIn
	{
		byte[i] = getc(fIn);
	}

    unsigned int width = *(int*)&byte[18];
	unsigned int height = *(int*)&byte[22];
	fwrite(byte, sizeof(unsigned char), 54, fOut);					//write the header back

	end = clock();
	walltime=(double)(((double)(end-start)* 1000)/(double)CLOCKS_PER_SEC);
    printf("Time to read & write header file for size : %d x %d is: %f miliseconds\n",height,width,walltime);

	printf("width: %d\n", width);
	printf("height: %d\n", height);

    int size = height*width;

	unsigned char* buffer = (unsigned char*)malloc(size * sizeof(unsigned char));
	unsigned char* out = (unsigned char*)malloc(size * sizeof(unsigned char));
	float* c = (float*)malloc(size * sizeof(float));

	start = clock();
	for (i = 0; i < height; i++)
	{
		for (j = 0; j < width; j++)
		{
			buffer[i * width + j] = getc(fIn);
		}
	}

    end = clock();
	walltime=(double)(((double)(end-start)* 1000)/(double)CLOCKS_PER_SEC);
    printf("Time to read image file into buffer for size : %d x %d is: %f miliseconds\n",height,width,walltime);


    start = clock();
	for (i = 0; i < size; i++)
	{
		c[i] = ((float)(buffer[i])) / (255.0f);
	}
	end = clock();
	walltime=(double)(((double)(end-start)* 1000)/(double)CLOCKS_PER_SEC);
    printf("Time to convert pixel values in 0-1 range for image size : %d x %d is: %f miliseconds\n",height,width,walltime);

    // Process image
    process_image(c, height, width);

	start = clock();
	for (i = 0; i < size; i++)
	{
		x = (int)(c[i]*255.0f);
		out[i] = (unsigned char)x;
	}

	fwrite(out, sizeof(unsigned char), size, fOut);           //write image data back to the file
    end = clock();
    walltime=(double)(((double)(end-start)* 1000)/(double)CLOCKS_PER_SEC);
    printf("Time to convert pixel values in range of 0-255 & write image in output for image size : %d x %d is: %f miliseconds\n",height,width,walltime);

	fclose(fIn);
	fclose(fOut);

    free(c);
    free(buffer);
    free(out);
	return 0;
}
